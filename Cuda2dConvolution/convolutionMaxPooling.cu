#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
//#include <opencv2/opencv.hpp>

#include <stdio.h>
#include <stdlib.h>
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#define STB_IMAGE_IMPLEMENTATION
#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "stb_image.h"
#include "stb_image_write.h"

#define BLOCK_SIZE 16


// convolution MaxPooling loop
__global__ void convMax(float* output, float* input, float* kernel, int input_width, int input_height)
{
    // 2D thread index
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    // Check if we are within the bounds of the input image
    if (x < input_width && y < input_height)
    {
        // Compute the output pixel value
        float sum = 0;
        int tempMax = 0;
        int Max = 0;
        for (int ky = 0; ky < 2; ++ky)
        {
            for (int kx = 0; kx < 2; ++kx)
            {
                // Compute the index of the input pixel
                int px = x - 1 + kx;
                int py = y - 1 + ky;

                // Check if the pixel is within the bounds of the input image
                if (px >= 0 && px < input_width && py >= 0 && py < input_height)
                {
                    // Perform the convolution
                    sum += input[py * input_width + px] * kernel[ky * 2 + kx];
                    if (input[py * input_width + px] > tempMax)
                    {
                      tempMax = input[py * input_width + px];
                    }
                    max = tempMax;
                }
            }
        }
        // Set the output pixel value
        output[y * input_width + x] = max;
    }
}


int main()
{
    // Load the input image
    int width, height, componentCount;
    //cv::Mat input_image = cv::imread("puppyGrey.png", cv::IMREAD_GRAYSCALE);
     unsigned char *input_image = stbi_load("puppyGrey.png", &width, &height, &componentCount, 4);
    //if (input_image.empty())
    //{
    //    std::cerr << "Failed to load input image." << std::endl;
    //    return 1;
    //}

    // Get the image dimensions
    int input_width = width;
    int input_height = height;

    // Allocate host memory for the input image, kernel, and output image
    float* h_input = new float[input_width * input_height];
    float* h_kernel = new float[2 * 2];
    float* h_output = new float[input_width * input_height];

    // Copy the input image data to the host
    for (int y = 0; y < input_height; ++y)
    {
        for (int x = 0; x < input_width; ++x)
        {
            h_input[y * input_width + x] = static_cast<float>(input_image[(y * input_width + x) * 4]);
        }
    }

    // Set the values of the kernel
    h_kernel[0] = 1; h_kernel[1] = 1; h_kernel[2] = 1;
    h_kernel[3] = 1;// h_kernel[4] = 0; h_kernel[5] = 0;
 //   h_kernel[6] = 1; h_kernel[7] = 1; h_kernel[8] = 1;

// Allocate device memory
float* d_input;
float* d_kernel;
float* d_output;
int input_size = input_width * input_height;
int kernel_size = 3 * 3;
int output_size = input_width * input_height;
hipMalloc((void**)&d_input, input_size * sizeof(float));
hipMalloc((void**)&d_kernel, kernel_size * sizeof(float));
hipMalloc((void**)&d_output, output_size * sizeof(float));

// Copy data from host to device
hipMemcpy(d_input, h_input, input_size * sizeof(float), hipMemcpyHostToDevice);
hipMemcpy(d_kernel, h_kernel, kernel_size * sizeof(float), hipMemcpyHostToDevice);

// Launch the kernel
dim3 block(BLOCK_SIZE, BLOCK_SIZE);
dim3 grid((input_width + BLOCK_SIZE - 1) / BLOCK_SIZE, (input_height + BLOCK_SIZE - 1) / BLOCK_SIZE);
conv2D<<<grid, block>>>(d_output, d_input, d_kernel, input_width, input_height);

// Copy data from device to host
hipMemcpy(h_output, d_output, output_size * sizeof(float), hipMemcpyDeviceToHost);

// Create the output image
//cv::Mat output_image(input_height, input_width, CV_8UC1);
//for (int y = 0; y < input_height; ++y)
//{
//    for (int x = 0; x < input_width; ++x)
//    {
//        output_image.at<uchar>(y, x) = static_cast<uchar>(h_output[y * input_width + x]);
//    }
//}

unsigned char* h_output_image = (unsigned char*)malloc(input_size * sizeof(unsigned char));
for (int y = 0; y < input_height; ++y)
{
    for (int x = 0; x < input_width; ++x)
    {
        h_output_image[y * input_width + x] = static_cast<unsigned char>(h_output[y * input_width + x]);
    }
}

// Save the output image
// const char *fileNameOut = "blue.png";

    // Write image back to disk
    printf("Writing png to disk...\r\n");
   // stbi_write_png(fileNameOut, width, height, 4, d_output, 4 * width);

    stbi_write_png("output_image.png", input_width, input_height, 1, h_output_image, input_width);


// Free host memory
delete[] h_input;
delete[] h_kernel;
delete[] h_output;

// Free device memory
hipFree(d_input);
hipFree(d_kernel);
hipFree(d_output);

return 0;
}


